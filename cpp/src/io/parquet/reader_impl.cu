#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO Parquet reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/gpuinflate.h>

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <algorithm>
#include <array>
#include <regex>

#include "printer.hpp"

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
// Import functionality that's independent of legacy code
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {

type_id to_type_id(SchemaElement const &schema,                   
                   bool strings_to_categorical,
                   type_id timestamp_type_id)
{
  parquet::Type physical = schema.type;
  parquet::ConvertedType logical = schema.converted_type;
  int32_t decimal_scale = schema.decimal_scale;

  // printf("PQT : %d / %d\n", (int)physical, (int)logical);

  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  switch (logical) {
    case parquet::UINT_8:
    case parquet::INT_8: return type_id::INT8;
    case parquet::UINT_16:
    case parquet::INT_16: return type_id::INT16;
    case parquet::DATE: return type_id::TIMESTAMP_DAYS;
    case parquet::TIMESTAMP_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MICROSECONDS;
    case parquet::TIMESTAMP_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MILLISECONDS;
    case parquet::DECIMAL:
      if (decimal_scale != 0 || (physical != parquet::INT32 && physical != parquet::INT64)) {
        return type_id::FLOAT64;
      }
      break;

    case parquet::LIST:
      return type_id::LIST;    

    default: break;
  }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN: return type_id::BOOL8;
    case parquet::INT32: return type_id::INT32;
    case parquet::INT64: return type_id::INT64;
    case parquet::FLOAT: return type_id::FLOAT32;
    case parquet::DOUBLE: return type_id::FLOAT64;
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to INT32 (32-bit hash) or STRING
      return strings_to_categorical ? type_id::INT32 : type_id::STRING;
    case parquet::INT96:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    default: break;
  }

  return type_id::EMPTY;
}


/**
 * @brief Function that translates cuDF time unit to Parquet clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_timestamp(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  printf("converted type : %d\n", converted);
  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64
  }
  return std::make_tuple(type_width, clock_rate, converted_type);
}

}  // namespace

/**
 * @brief Class for parsing dataset metadata
 */
struct metadata : public FileMetaData {
  explicit metadata(datasource *source)
  {
    constexpr auto header_len = sizeof(file_header_s);
    constexpr auto ender_len  = sizeof(file_ender_s);

    const auto len           = source->size();
    const auto header_buffer = source->host_read(0, header_len);
    const auto header        = (const file_header_s *)header_buffer->data();
    const auto ender_buffer  = source->host_read(len - ender_len, ender_len);
    const auto ender         = (const file_ender_s *)ender_buffer->data();
    CUDF_EXPECTS(len > header_len + ender_len, "Incorrect data source");
    CUDF_EXPECTS(header->magic == PARQUET_MAGIC && ender->magic == PARQUET_MAGIC,
                 "Corrupted header or footer");
    CUDF_EXPECTS(ender->footer_len != 0 && ender->footer_len <= (len - header_len - ender_len),
                 "Incorrect footer length");

    const auto buffer = source->host_read(len - ender->footer_len - ender_len, ender->footer_len);
    CompactProtocolReader cp(buffer->data(), ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot initialize schema");
  }

  inline int64_t get_total_rows() const { return num_rows; }
  inline int get_num_row_groups() const { return row_groups.size(); }
  inline int get_num_columns() const { return row_groups[0].columns.size(); }

  inline SchemaElement const& get_column_schema(int col_index) const
  {
    return schema[row_groups[0].columns[col_index].schema_idx];
  }

  inline int get_column_leaf_schema_index(int col_index) const
  {
    return row_groups[0].columns[col_index].leaf_schema_idx;
  }

  inline SchemaElement const& get_column_leaf_schema(int col_index) const
  {
    return schema[get_column_leaf_schema_index(col_index)];
  }
  
  std::string get_column_name(const std::vector<std::string> &path_in_schema)
  {
    std::string s = (path_in_schema.size() > 0) ? path_in_schema[0] : "";
    for (size_t i = 1; i < path_in_schema.size(); i++) { s += "." + path_in_schema[i]; }
    return s;
  }
    
  std::string get_column_name(int schema_idx)
  {    
    return schema[schema_idx].name;
  }  

  std::vector<std::string> get_column_names()
  {
    std::vector<std::string> all_names;
    if (row_groups.size() != 0) {
      for (const auto &chunk : row_groups[0].columns) {
        all_names.emplace_back(get_column_name(chunk.meta_data.path_in_schema));
        // all_names.emplace_back(get_column_name(chunk.schema_idx));
      }
    }
    return all_names;
  }

  /**
   * @brief Extracts the pandas "index_columns" section
   *
   * PANDAS adds its own metadata to the key_value section when writing out the
   * dataframe to a file to aid in exact reconstruction. The JSON-formatted
   * metadata contains the index column(s) and PANDA-specific datatypes.
   *
   * @return comma-separated index column names in quotes
   */
  std::string get_pandas_index()
  {
    auto it = std::find_if(key_value_metadata.begin(),
                           key_value_metadata.end(),
                           [](const auto &item) { return item.key == "pandas"; });
    if (it != key_value_metadata.end()) {
      // Captures a list of quoted strings found inside square brackets after `"index_columns":`
      // Inside quotes supports newlines, brackets, escaped quotes, etc.
      // One-liner regex:
      // "index_columns"\s*:\s*\[\s*((?:"(?:|(?:.*?(?![^\\]")).?)[^\\]?",?\s*)*)\]
      // Documented below.
      std::regex index_columns_expr{
        R"("index_columns"\s*:\s*\[\s*)"  // match preamble, opening square bracket, whitespace
        R"(()"                            // Open first capturing group
        R"((?:")"                         // Open non-capturing group match opening quote
        R"((?:|(?:.*?(?![^\\]")).?))"     // match empty string or anything between quotes
        R"([^\\]?")"                      // Match closing non-escaped quote
        R"(,?\s*)"                        // Match optional comma and whitespace
        R"()*)"                           // Close non-capturing group and repeat 0 or more times
        R"())"                            // Close first capturing group
        R"(\])"                           // Match closing square brackets
      };
      std::smatch sm;
      if (std::regex_search(it->value, sm, index_columns_expr)) { return std::move(sm[1].str()); }
    }
    return "";
  }

  /**
   * @brief Extracts the column name(s) used for the row indexes in a dataframe
   *
   * @param names List of column names to load, where index column name(s) will be added
   */
  void add_pandas_index_names(std::vector<std::string> &names)
  {
    auto str = get_pandas_index();
    if (str.length() != 0) {
      std::regex index_name_expr{R"(\"((?:\\.|[^\"])*)\")"};
      std::smatch sm;
      while (std::regex_search(str, sm, index_name_expr)) {
        if (sm.size() == 2) {  // 2 = whole match, first item
          if (std::find(names.begin(), names.end(), sm[1].str()) == names.end()) {
            std::regex esc_quote{R"(\\")"};
            names.emplace_back(std::move(std::regex_replace(sm[1].str(), esc_quote, R"(")")));
          }
        }
        str = sm.suffix();
      }
    }
  }

  /**
   * @brief Filters and reduces down to a selection of row groups
   *
   * @param row_group Index of the row group to select
   * @param max_rowgroup_count Max number of consecutive row groups if > 0
   * @param row_group_indices Arbitrary rowgroup list[max_rowgroup_count] if non-null
   * @param row_start Starting row of the selection
   * @param row_count Total number of rows selected
   *
   * @return List of row group indexes and its starting row
   */
  auto select_row_groups(size_type row_group,
                         size_type max_rowgroup_count,
                         const size_type *row_group_indices,
                         size_type &row_start,
                         size_type &row_count)
  {
    std::vector<std::pair<size_type, size_t>> selection;

    if (row_group_indices) {
      row_count = 0;
      for (size_type i = 0; i < max_rowgroup_count; i++) {
        auto rowgroup_idx = row_group_indices[i];
        CUDF_EXPECTS(rowgroup_idx >= 0 && rowgroup_idx < get_num_row_groups(),
                     "Invalid rowgroup index");
        selection.emplace_back(rowgroup_idx, row_count);
        row_count += row_groups[rowgroup_idx].num_rows;
      }
    } else if (row_group != -1) {
      CUDF_EXPECTS(row_group < get_num_row_groups(), "Non-existent row group");
      row_count = 0;
      do {
        selection.emplace_back(row_group, row_start + row_count);
        row_count += row_groups[row_group].num_rows;
      } while (--max_rowgroup_count > 0 && ++row_group < get_num_row_groups());
    } else {
      row_start = std::max(row_start, 0);
      if (row_count < 0) {
        row_count = static_cast<size_type>(
          std::min<int64_t>(get_total_rows(), std::numeric_limits<size_type>::max()));
      }
      CUDF_EXPECTS(row_count >= 0, "Invalid row count");
      CUDF_EXPECTS(row_start <= get_total_rows(), "Invalid row start");

      for (size_t i = 0, count = 0; i < row_groups.size(); ++i) {
        size_t chunk_start_row = count;
        count += row_groups[i].num_rows;
        if (count > static_cast<size_t>(row_start) || count == 0) {
          selection.emplace_back(i, chunk_start_row);
        }
        if (count >= static_cast<size_t>(row_start) + static_cast<size_t>(row_count)) { break; }
      }
    }

    return selection;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param include_index Whether to always include the PANDAS index column(s)
   *
   * @return List of column names
   */
  auto select_columns(std::vector<std::string> use_names, bool include_index)
  {
    std::vector<std::pair<int, std::string>> selection;

    const auto names = get_column_names();
    if (use_names.empty()) {
      // No columns specified; include all in the dataset
      for (const auto &name : names) { selection.emplace_back(selection.size(), name); }
    } else {
      // Load subset of columns; include PANDAS index unless excluded
      if (include_index) { add_pandas_index_names(use_names); }
      for (const auto &use_name : use_names) {
        for (size_t i = 0; i < names.size(); ++i) {
          PRINTF("NAME : %s\n", names[i].c_str());

          if (names[i] == use_name) {
            selection.emplace_back(i, names[i]);
            break;
          }
        }
      }
    }

    return selection;
  }
};

void reader::impl::read_column_chunks(std::vector<rmm::device_buffer> &page_data,
                                      hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                      size_t begin_chunk,
                                      size_t end_chunk,
                                      const std::vector<size_t> &column_chunk_offsets,
                                      hipStream_t stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    const size_t io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    const bool is_compressed = (chunks[chunk].codec != parquet::Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      const size_t next_offset = column_chunk_offsets[next_chunk];
      const bool is_next_compressed =
        (chunks[next_chunk].codec != parquet::Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto buffer         = _source->host_read(io_offset, io_size);
      page_data[chunk]    = rmm::device_buffer(buffer->data(), buffer->size(), stream);
      uint8_t *d_compdata = reinterpret_cast<uint8_t *>(page_data[chunk].data());
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
}

size_t reader::impl::count_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                        hipStream_t stream)
{
  size_t total_pages = 0;

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(
    chunks.host_ptr(), chunks.device_ptr(), chunks.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

void reader::impl::decode_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                       hostdevice_vector<gpu::PageInfo> &pages,
                                       hipStream_t stream)
{
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info     = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(
    pages.host_ptr(), pages.device_ptr(), pages.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));
}

rmm::device_buffer reader::impl::decompress_page_data(
  hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
  hostdevice_vector<gpu::PageInfo> &pages,
  hipStream_t stream)
{
  auto for_each_codec_page = [&](parquet::Compression codec, const std::function<void(size_t)> &f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) { f(page_count + k); }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_vector<uint8_t> debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;
  std::array<std::pair<parquet::Compression, size_t>, 3> codecs{std::make_pair(parquet::GZIP, 0),
                                                                std::make_pair(parquet::SNAPPY, 0),
                                                                std::make_pair(parquet::BROTLI, 0)};

  for (auto &codec : codecs) {
    for_each_codec_page(codec.first, [&](size_t page) {
      total_decomp_size += pages[page].uncompressed_page_size;
      codec.second++;
      num_comp_pages++;
    });
    if (codec.first == parquet::BROTLI && codec.second > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.second));
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, stream);
  hostdevice_vector<gpu_inflate_input_s> inflate_in(0, num_comp_pages, stream);
  hostdevice_vector<gpu_inflate_status_s> inflate_out(0, num_comp_pages, stream);

  size_t decomp_offset = 0;
  int32_t argc         = 0;
  for (const auto &codec : codecs) {
    if (codec.second > 0) {
      int32_t start_pos = argc;

      for_each_codec_page(codec.first, [&](size_t page) {
        auto dst_base              = static_cast<uint8_t *>(decomp_pages.data());
        inflate_in[argc].srcDevice = pages[page].page_data;
        inflate_in[argc].srcSize   = pages[page].compressed_page_size;
        inflate_in[argc].dstDevice = dst_base + decomp_offset;
        inflate_in[argc].dstSize   = pages[page].uncompressed_page_size;

        inflate_out[argc].bytes_written = 0;
        inflate_out[argc].status        = static_cast<uint32_t>(-1000);
        inflate_out[argc].reserved      = 0;

        pages[page].page_data = (uint8_t *)inflate_in[argc].dstDevice;
        decomp_offset += inflate_in[argc].dstSize;
        argc++;
      });

      CUDA_TRY(hipMemcpyAsync(inflate_in.device_ptr(start_pos),
                               inflate_in.host_ptr(start_pos),
                               sizeof(decltype(inflate_in)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      CUDA_TRY(hipMemcpyAsync(inflate_out.device_ptr(start_pos),
                               inflate_out.host_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      switch (codec.first) {
        case parquet::GZIP:
          CUDA_TRY(gpuinflate(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              1,
                              stream))
          break;
        case parquet::SNAPPY:
          CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              stream));
          break;
        case parquet::BROTLI:
          CUDA_TRY(gpu_debrotli(inflate_in.device_ptr(start_pos),
                                inflate_out.device_ptr(start_pos),
                                debrotli_scratch.data().get(),
                                debrotli_scratch.size(),
                                argc - start_pos,
                                stream));
          break;
        default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
      }
      CUDA_TRY(hipMemcpyAsync(inflate_out.host_ptr(start_pos),
                               inflate_out.device_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyDeviceToHost,
                               stream));
    }
  }
  CUDA_TRY(hipStreamSynchronize(stream));

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  CUDA_TRY(hipMemcpyAsync(
    pages.device_ptr(), pages.host_ptr(), pages.memory_size(), hipMemcpyHostToDevice, stream));

  return decomp_pages;
}

void reader::impl::preprocess_nested_columns(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                      hostdevice_vector<gpu::PageInfo> &pages,
                                      size_t min_row,
                                      size_t total_rows,
                                      hipStream_t stream)
{
  // for nested columns, the # of rows in the metadata is not sufficient to determine
  // the size of the outgoing columns.  we need to parse the repetition and definition
  // levels to determine this
  printf("PREPROCESS START\n");
  CUDA_TRY(gpu::PreprocessNestingData(pages.device_ptr(),
                                pages.size(),
                                chunks.device_ptr(),
                                chunks.size(),
                                total_rows,
                                min_row,
                                stream));
  
  CUDA_TRY(hipStreamSynchronize(stream));  
  printf("PREPROCESS END\n");
}

void reader::impl::decode_page_data(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                    hostdevice_vector<gpu::PageInfo> &pages,
                                    size_t min_row,
                                    size_t total_rows,
                                    const std::vector<int> &chunk_map,
                                    std::vector<column_buffer> &out_buffers,
                                    hipStream_t stream)
{  
  printf("DECODE START\n");

  auto is_dict_chunk = [](const gpu::ColumnChunkDesc &chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) { total_str_dict_indexes += pages[page_count].num_values; }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  rmm::device_vector<gpu::nvstrdesc_s> str_dict_index;
  if (total_str_dict_indexes > 0) { str_dict_index.resize(total_str_dict_indexes); }

  // Update chunks with pointers to column data
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data().get() + str_ofs;
      str_ofs += pages[page_count].num_values;
    }

    int output_depth = chunks[c].max_level[gpu::level_type::REPETITION];    
    printf("OD : %d\n", output_depth);
    
    // setup base pointers. need to do this better
    size_type buf_bytes = sizeof(void*) * (output_depth+1);
    hipMalloc(&chunks[c].valid_map_base, buf_bytes);
    hipMalloc(&chunks[c].column_data_base, buf_bytes);
    std::vector<uint32_t*> valids(output_depth+1);
    std::vector<void*> data(output_depth+1);
    column_buffer* buf = &out_buffers[chunk_map[c]];
    for(int idx=0; idx<=output_depth; idx++){      
      data[idx] = buf->data();             
      valids[idx] = buf->null_mask();
      if(idx < output_depth){
        printf("BPI : %d\n", idx);
        CUDF_EXPECTS(buf->children.size() > 0, "Encountered a malformed column_buffer");
        buf = &buf->children[0];
      }
    }
    printf("H %lu %lu\n", (uint64_t)valids[0], (uint64_t)data[0]);
    printf("H %lu %lu\n", (uint64_t)valids[1], (uint64_t)data[1]);
    hipMemcpy(chunks[c].valid_map_base, valids.data(), buf_bytes, hipMemcpyHostToDevice);
    hipMemcpy(chunks[c].column_data_base, data.data(), buf_bytes, hipMemcpyHostToDevice);

    // column_data_base will always point to leaf data, even for nested types.    

    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  if (total_str_dict_indexes > 0) {
    //CUDA_TRY(gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), stream));
  }  

  printf("DecodePageData : pages size : %lu, chunks size : %lu, total rows : %lu, min row : %lu\n",
        pages.size(), chunks.size(), total_rows, min_row);
  CUDA_TRY(gpu::DecodePageData(pages.device_ptr(),
                               pages.size(),
                               chunks.device_ptr(),
                               chunks.size(),
                               total_rows,
                               min_row,
                               stream));
  CUDA_TRY(hipMemcpyAsync(
    pages.host_ptr(), pages.device_ptr(), pages.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  // ROW VALUE
  /*
  for (size_t i = 0; i < pages.size(); i++) {
    if (pages[i].num_rows > 0) {
      const size_t c = pages[i].chunk_idx;
      if (c < chunks.size()) {
        // TODO
        // out_buffers[chunk_map[c]].null_count() += pages[i].num_rows - pages[i].valid_count;
      }
    }
  }
  */

  printf("DECODE END\n");
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _source(std::move(source)), _mr(mr)
{
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<metadata>(_source.get());

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(options.columns, options.use_pandas_metadata);

  // Override output timestamp resolution if requested
  if (options.timestamp_type.id() != EMPTY) { _timestamp_type = options.timestamp_type; }

  // Strings may be returned as either string or categorical columns
  _strings_to_categorical = options.strings_to_categorical;
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       size_type row_group,
                                       size_type max_rowgroup_count,
                                       const size_type *row_group_indices,
                                       hipStream_t stream)
{
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata out_metadata;

  // Select only row groups required
  const auto selected_row_groups = _metadata->select_row_groups(
    row_group, max_rowgroup_count, row_group_indices, skip_rows, num_rows);

  // Get a list of column data types
  std::vector<data_type> column_types;
  if (_metadata->row_groups.size() != 0) {
    for (const auto &col : _selected_columns) {
      auto &col_schema = _metadata->get_column_schema(col.first);
      auto col_type    = to_type_id(col_schema,
                                    _strings_to_categorical,
                                    _timestamp_type.id());
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      column_types.emplace_back(col_type);
    }
  }
  printf("Selected %lu columns\n", column_types.size());
  out_columns.reserve(column_types.size());
  
  if (selected_row_groups.size() != 0 && column_types.size() != 0) {
    // Descriptors for all the chunks that make up the selected columns
    const auto num_columns = _selected_columns.size();
    const auto num_chunks  = selected_row_groups.size() * num_columns;
    hostdevice_vector<gpu::ColumnChunkDesc> chunks(0, num_chunks, stream);

    // Association between each column chunk and its column
    std::vector<int> chunk_map(num_chunks);

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> page_data(num_chunks);

    // Keep track of column chunk file offsets
    std::vector<size_t> column_chunk_offsets(num_chunks);

    // column nullability
    std::vector<std::vector<bool>> col_nullability(column_types.size());

    // information needed allocate columns (including potential nesting)
    bool has_nesting = false;
    /*
    std::vector<size_type*> g_col_sizes_nested(column_types.size());
    std::vector<size_type*> h_col_sizes_nested(column_types.size());
    std::vector<size_type*> g_col_remap_nested(column_types.size());    
    std::vector<size_type*> h_col_remap_nested(column_types.size());
    std::vector<size_type*> g_col_valid_count_nested(column_types.size());    
    std::vector<size_type*> h_col_remap_nested(column_types.size());
    */
    // nesting information per column
    std::vector<hostdevice_vector<gpu::ColumnNestingInfo>> nested_column_info(column_types.size()); 
    for (const auto &col : _selected_columns) {
      // for non-nested columns, these will be the same. for lists, the leaf schema represents the bottom
      // level type (int, string, etc).  For future types like structs, this will probably get more complicated.
      auto &col_schema = _metadata->get_column_schema(col.first);       
      auto &leaf_schema = _metadata->get_column_leaf_schema(col.first);       

      // depth of the nesting in the output (1 == no nesting, 2 == 1 level, etc)
      int output_depth = (leaf_schema.max_repetition_level + 1);

      col_nullability[col.first] = std::move(std::vector<bool>(output_depth));
      
      // for non-nested columns, just set nullability
      if(leaf_schema.max_repetition_level == 0){
        // i'm not sure this is exactly correct, but it's how we we've been doing it so far.
        // seems like we should be checking the repetition type (required / optional)
        col_nullability[col.first][0] = leaf_schema.max_definition_level != 0 ? true : false;        
      } 
      // for nested columns:
      // - allocate output space for computing column sizes (which will be done on the gpu)
      // - build mapping of schema definition levels to real cudf child column output indices
      // - fill in nullability
      else {
        size_t size = output_depth + (leaf_schema.max_definition_level + 1);
        printf("calloc size : %lu, %lu\n", size, nested_column_info.size());        
        nested_column_info[col.first] = std::move(hostdevice_vector<gpu::ColumnNestingInfo>(size, size, stream));

        // zero data
        hipMemset(nested_column_info[col.first].host_ptr(), 0, sizeof(gpu::ColumnNestingInfo) * size);
        
        // fill in host-side schema-remapping        
        // printf("remap : %lu\n", (uint64_t)remap);
        // int schema_idx = _metadata->row_groups[0].columns[col.first].schema_idx;
        int schema_idx = _metadata->get_column_leaf_schema_index(col.first);
        int output_col_idx = leaf_schema.max_repetition_level;
        while(schema_idx > 0){
          printf("Schema : %d, %d, %d, %d, %d\n", schema_idx, 
            _metadata->schema[schema_idx].type,
            _metadata->schema[schema_idx].repetition_type,
            _metadata->schema[schema_idx].max_definition_level,
            _metadata->schema[schema_idx].max_repetition_level);          

          // this field is misnamed. it really should just be "definition level"
          int d = _metadata->schema[schema_idx].max_definition_level;

          // the list definition itself
          if(_metadata->schema[schema_idx].repetition_type == REPEATED){
            nested_column_info[col.first][d].remap = output_col_idx | cudf::io::parquet::gpu::REPEATED_FIELD_BIT;
            output_col_idx--;            
          }
          // the element within 
          else {
            nested_column_info[col.first][d].remap = output_col_idx;
            col_nullability[col.first][output_col_idx] = _metadata->schema[schema_idx].repetition_type == OPTIONAL ? true : false;
          }
          schema_idx = _metadata->schema[schema_idx].parent_idx;          
        }

        for(int s_idx=0; s_idx<=leaf_schema.max_definition_level; s_idx++){
          printf("hcols[%d] : 0x%x\n", s_idx, nested_column_info[col.first][s_idx].remap);
        }
        for(int s_idx=0; s_idx<output_depth; s_idx++){
          printf("nullable[%d] : %s\n", s_idx, col_nullability[col.first][s_idx] ? "yes" : "no");
        }

        // copy to device
        nested_column_info[col.first].host_to_device(stream);
        has_nesting = true;
      }
    }

    // Initialize column chunk information
    size_t total_decompressed_size = 0;
    auto remaining_rows            = num_rows;
    for (const auto &rg : selected_row_groups) {
      const auto &row_group = _metadata->row_groups[rg.first];
      auto row_group_start  = rg.second;
      auto row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
      auto io_chunk_idx     = chunks.size();

      for (size_t i = 0; i < num_columns; ++i) {
        auto col         = _selected_columns[i];
        auto &col_meta   = row_group.columns[col.first].meta_data;

        // is it safe to assume the schema is always the same across row groups? I think so?
        // _metadata->schema[row_group.columns[col.first].schema_idx];
        
        // the root schema (which in the case of nested types is different from the leaf schema).
        // the # of rows in the row group is relative to the root
        auto &root_schema = _metadata->get_column_schema(col.first);        
        // the leaf schema represents the -values- encoded in the data, which in the case
        // of nested types, is different from the # of rows
        auto &leaf_schema = _metadata->get_column_leaf_schema(col.first);                

        // Spec requires each row group to contain exactly one chunk for every
        // column. If there are too many or too few, continue with best effort
        // if (col.second != _metadata->get_column_name(col_schema.schema_idx)) {
        /*
        if (col.second != _metadata->get_column_name(row_group.columns[col.first].schema_idx)) {
          std::cerr << "Detected mismatched column chunk" << std::endl;
          continue;
        }
        */
        if (chunks.size() >= chunks.max_size()) {
          std::cerr << "Detected too many column chunks" << std::endl;
          continue;
        }

        int32_t type_width;
        int32_t clock_rate;
        int8_t converted_type;
        std::tie(type_width, clock_rate, converted_type) =
          conversion_info(column_types[i].id(),
                          _timestamp_type.id(),
                          leaf_schema.type,
                          leaf_schema.converted_type,
                          leaf_schema.type_length);

        column_chunk_offsets[chunks.size()] =
          (col_meta.dictionary_page_offset != 0)
            ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
            : col_meta.data_page_offset;        

        // printf("CHUNK INSERT : %d, %d, %lu %d, %d\n", col_schema.type, converted_type, col_meta.num_values, col_schema.max_definition_level, col_schema.max_repetition_level);        

        bool is_nested = cudf::is_nested(column_types[i]);
        printf("NESTED : %s\n", is_nested ? "yes" : "no");
        printf("Root type : %d %d\n", root_schema.type, root_schema.converted_type);
        printf("Leaf type : %d %d\n", leaf_schema.type, leaf_schema.converted_type);
        printf("R / V : %d, %d\n", static_cast<int>(row_group.num_rows), static_cast<int>(col_meta.num_values));

        chunks.insert(gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                                           nullptr,
                                           col_meta.num_values,
                                           leaf_schema.type,
                                           type_width,
                                           row_group_start,
                                           row_group_rows,
                                           leaf_schema.max_definition_level,
                                           leaf_schema.max_repetition_level,
                                           required_bits(leaf_schema.max_definition_level),
                                           required_bits(leaf_schema.max_repetition_level),
                                           col_meta.codec,
                                           converted_type,
                                           leaf_schema.decimal_scale,
                                           clock_rate,
                                           i,
                                           is_nested ? nested_column_info[i].device_ptr() : nullptr));

        // Map each column chunk to its column index
        chunk_map[chunks.size() - 1] = i;

        if (col_meta.codec != Compression::UNCOMPRESSED) {
          total_decompressed_size += col_meta.total_uncompressed_size;
        }
      }
      // Read compressed chunk data to device memory
      read_column_chunks(
        page_data, chunks, io_chunk_idx, chunks.size(), column_chunk_offsets, stream);

      remaining_rows -= row_group.num_rows;
    }
    assert(remaining_rows <= 0);

    // Process dataset chunk pages into output columns
    const auto total_pages = count_page_headers(chunks, stream);
    if (total_pages > 0) {
      hostdevice_vector<gpu::PageInfo> pages(total_pages, total_pages, stream);
      rmm::device_buffer decomp_page_data;

      // decoding of column/page information
      decode_page_headers(chunks, pages, stream);
      if (total_decompressed_size > 0) {
        decomp_page_data = decompress_page_data(chunks, pages, stream);
        // Free compressed data
        for (size_t c = 0; c < chunks.size(); c++) {
          if (chunks[c].codec != parquet::Compression::UNCOMPRESSED && page_data[c].size() != 0) {
            page_data[c].resize(0);
            page_data[c].shrink_to_fit();
          }
        }
      }

      // if we have any nested columns, preprocess them now
      if(has_nesting){
        preprocess_nested_columns(chunks, pages, skip_rows, num_rows, stream);
      }

      std::vector<column_buffer> out_buffers;
      out_buffers.reserve(column_types.size());
      for (size_t i = 0; i < column_types.size(); ++i) {
        auto col = _selected_columns[i];
        auto &leaf_schema = _metadata->get_column_leaf_schema(col.first);

        int output_depth = leaf_schema.max_repetition_level + 1;

        // nested.  for now, we are interpreting this to mean LIST but ultimately
        // all nested types will probably go through this path
        if(output_depth > 1){
          // retrieve computed info back to the host
          nested_column_info[i].device_to_host(stream);
          printf("A\n"); 

          // the root buffer
          printf("OBR : %d\n", column_types[i].id());
          out_buffers.emplace_back(column_buffer{column_types[i], nested_column_info[i][0].size+1, col_nullability[i][0], stream, _mr});
          column_buffer* col = &out_buffers[out_buffers.size()-1];          
          // nested buffers
          for(int idx=1; idx<output_depth-1; idx++){            
            // note : all levels in a list column besides the leaf are offsets, so their length is always +1
            printf("OBC : %d\n", column_types[i].id());
            col->children.push_back(column_buffer{column_types[i], nested_column_info[i][idx].size+1,
                                    col_nullability[i][idx], stream, _mr});            
            col = &col->children[0];                        
          }                    
          // leaf buffer. note : the leaf type is not "LIST". it is a plain data type. 
          printf("OBL : %d\n", to_type_id(leaf_schema, _strings_to_categorical, _timestamp_type.id()));
          col->children.push_back(column_buffer{data_type{to_type_id(leaf_schema, _strings_to_categorical, _timestamp_type.id())},
                        nested_column_info[i][output_depth-1].size, col_nullability[i][output_depth-1], stream, _mr});
        }
        // other types
        else {
          // note : num_rows == # values for non-nested types
          out_buffers.emplace_back(column_buffer{column_types[i], num_rows, col_nullability[i][0], stream, _mr});
        }
      }

      // decoding of column data itself      
      decode_page_data(chunks, pages, skip_rows, num_rows, chunk_map, out_buffers, stream);
            
      for (size_t i = 0; i < column_types.size(); ++i) {        
        // retrieve validity counts
        nested_column_info[i].device_to_host(stream);

        auto col = _selected_columns[i];
        auto &leaf_schema = _metadata->get_column_leaf_schema(col.first);

        int output_depth = leaf_schema.max_repetition_level + 1;
                
        uint8_t buf[512];
        column_buffer* cb = &out_buffers[i];
        for(int idx=0; idx<output_depth-1; idx++){          
          cb->_null_count = nested_column_info[i][idx].null_count;
          hipMemcpy(buf, cb->data(), cb->size * 4, hipMemcpyDeviceToHost);
          printf("offsets, depth %d : \n", idx);
          for(int s_idx=0; s_idx<cb->size; s_idx++){
            printf("%d, ", ((int*)buf)[s_idx]);
          }
          printf("\n");          
          cb = &cb->children[0];
        }
        {
          cb->_null_count = nested_column_info[i][output_depth-1].null_count;
          hipMemcpy(buf, cb->data(), cb->size * 4, hipMemcpyDeviceToHost);
          printf("vals, depth %d : \n", output_depth-1);
          for(int s_idx=0; s_idx<cb->size; s_idx++){
            printf("%d, ", ((int*)buf)[s_idx]);
          }
          printf("\n");
        }         
        
        printf("OBE : %d %d\n", out_buffers[0].type.id(), out_buffers[0].children[0].type.id());
        out_columns.emplace_back(make_column(out_buffers[i], stream, _mr));
      }      
    }
  }

  PRINTF("STAGE 3\n");

  // Create empty columns as needed
  for (size_t i = out_columns.size(); i < column_types.size(); ++i) {
    out_columns.emplace_back(make_empty_column(column_types[i]));
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_selected_columns.size());
  for (size_t i = 0; i < _selected_columns.size(); i++) {
    out_metadata.column_names[i] = _selected_columns[i].second;
  }
  // Return user metadata
  for (const auto &kv : _metadata->key_value_metadata) {
    out_metadata.user_data.insert({kv.key, kv.value});
  }

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::string filepath,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(datasource::create(filepath), options, mr))
{
}

// Forward to implementation
reader::reader(std::unique_ptr<cudf::io::datasource> source,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(std::move(source), options, mr))
{
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read_all(hipStream_t stream)
{
  return _impl->read(0, -1, -1, -1, nullptr, stream);
}

// Forward to implementation
table_with_metadata reader::read_row_group(size_type row_group,
                                           size_type row_group_count,
                                           hipStream_t stream)
{
  return _impl->read(0, -1, row_group, row_group_count, nullptr, stream);
}

// Forward to implementation
table_with_metadata reader::read_row_groups(const std::vector<size_type> &row_group_list,
                                            hipStream_t stream)
{
  return _impl->read(
    0, -1, -1, static_cast<size_type>(row_group_list.size()), row_group_list.data(), stream);
}

// Forward to implementation
table_with_metadata reader::read_rows(size_type skip_rows, size_type num_rows, hipStream_t stream)
{
  return _impl->read(skip_rows, (num_rows != 0) ? num_rows : -1, -1, -1, nullptr, stream);
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
