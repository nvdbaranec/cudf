#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scan.h>
#include <thrust/tuple.h>
#include <cudf/detail/utilities/release_assert.cuh>
#include <cudf/utilities/bit.hpp>
#include <io/utilities/block_utils.cuh>

#include "parquet_gpu.h"
#include "printer.hpp"

#define LOG2_NTHREADS (5 + 2)
#define NTHREADS (1 << LOG2_NTHREADS)
#define NZ_BFRSZ (NTHREADS * 2)

inline __device__ uint32_t rotl32(uint32_t x, uint32_t r)
{
  return __funnelshift_l(x, x, r);  // (x << r) | (x >> (32 - r));
};

inline __device__ int rolling_index(int index) { return index & (NZ_BFRSZ - 1); }

constexpr int column_watch = 4;

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {

struct page_state_s {
  const uint8_t *lvl_start[2];  // [def,rep]
  const uint8_t *data_start;
  const uint8_t *data_end;
  const uint8_t *dict_base;   // ptr to dictionary page data
  int32_t dict_size;          // size of dictionary data

  int32_t first_row;          // First row in page to output
  int32_t num_rows;           // Rows in page to decode (including rows to be skipped)
  
  int32_t first_output_value; // First value in page to output
  int32_t num_values;         // total # of values in the page

  int32_t dtype_len;          // Output data type length
  int32_t dtype_len_in;         // Can be larger than dtype_len if truncating 32-bit into 8-bit
  int32_t dict_bits;             // # of bits to store dictionary indices
  uint32_t dict_run;
  int32_t dict_val;
  uint32_t initial_rle_run[2];   // [def,rep]
  int32_t initial_rle_value[2];  // [def,rep]
  int32_t error;
  PageInfo page;
  ColumnChunkDesc col;

  // only used in the preprocess step
  //uint32_t **valid_map;
  //int32_t *valid_map_offset;

  // value decoding
  //uint8_t **data_out;
  int32_t nz_count;  // number of valid entries in nz_idx (write position in circular buffer)
  int32_t dict_pos;  // write position of dictionary indices
  int32_t out_pos;   // read position of final output
  int32_t ts_scale;  // timestamp scale: <0: divide by -ts_scale, >0: multiply by ts_scale
  uint32_t nz_idx[NZ_BFRSZ];    // circular buffer of non-null value positions
  uint32_t dict_idx[NZ_BFRSZ];  // Dictionary index, boolean, or string offset values
  uint32_t str_len[NZ_BFRSZ];   // String length for plain encoding of strings

  // level decoding
  int page_value_count;
  uint32_t rep[NZ_BFRSZ];
  uint32_t def[NZ_BFRSZ];
  int32_t lvl_count[2];
  int32_t rep_count;
  int32_t def_count;  
};

/**
 * @brief Computes a 32-bit hash when given a byte stream and range.
 *
 * MurmurHash3_32 implementation from
 * https://github.com/aappleby/smhasher/blob/master/src/MurmurHash3.cpp
 *
 * MurmurHash3 was written by Austin Appleby, and is placed in the public
 * domain. The author hereby disclaims copyright to this source code.
 *
 * @param[in] key The input data to hash
 * @param[in] len The length of the input data
 * @param[in] seed An initialization value
 *
 * @return The hash value
 */
__device__ uint32_t device_str2hash32(const char *key, size_t len, uint32_t seed = 33)
{
  const uint8_t *p  = reinterpret_cast<const uint8_t *>(key);
  uint32_t h1       = seed, k1;
  const uint32_t c1 = 0xcc9e2d51;
  const uint32_t c2 = 0x1b873593;
  int l             = len;
  // body
  while (l >= 4) {
    k1 = p[0] | (p[1] << 8) | (p[2] << 16) | (p[3] << 24);
    k1 *= c1;
    k1 = rotl32(k1, 15);
    k1 *= c2;
    h1 ^= k1;
    h1 = rotl32(h1, 13);
    h1 = h1 * 5 + 0xe6546b64;
    p += 4;
    l -= 4;
  }
  // tail
  k1 = 0;
  switch (l) {
    case 3: k1 ^= p[2] << 16;
    case 2: k1 ^= p[1] << 8;
    case 1:
      k1 ^= p[0];
      k1 *= c1;
      k1 = rotl32(k1, 15);
      k1 *= c2;
      h1 ^= k1;
  }
  // finalization
  h1 ^= len;
  h1 ^= h1 >> 16;
  h1 *= 0x85ebca6b;
  h1 ^= h1 >> 13;
  h1 *= 0xc2b2ae35;
  h1 ^= h1 >> 16;
  return h1;
}

/**
 * @brief Read a 32-bit varint integer
 *
 * @param[in,out] cur The current data position, updated after the read
 * @param[in] end The end data position
 *
 * @return The 32-bit value read
 */
inline __device__ uint32_t get_vlq32(const uint8_t *&cur, const uint8_t *end)
{
  uint32_t v = *cur++;
  if (v >= 0x80 && cur < end) {
    v = (v & 0x7f) | ((*cur++) << 7);
    if (v >= (0x80 << 7) && cur < end) {
      v = (v & ((0x7f << 7) | 0x7f)) | ((*cur++) << 14);
      if (v >= (0x80 << 14) && cur < end) {
        v = (v & ((0x7f << 14) | (0x7f << 7) | 0x7f)) | ((*cur++) << 21);
        if (v >= (0x80 << 21) && cur < end) {
          v = (v & ((0x7f << 21) | (0x7f << 14) | (0x7f << 7) | 0x7f)) | ((*cur++) << 28);
        }
      }
    }
  }
  return v;
}

/**
 * @brief Parse the beginning of the level section (definition or repetition),
 * initializes the initial RLE run & value, and returns the section length
 *
 * @param[in,out] s The page state
 * @param[in] cur The current data position
 * @param[in] end The end of the data
 * @param[in] level_bits The bits required

 */
__device__ uint32_t InitLevelSection(page_state_s *s,
                                     const uint8_t *cur,
                                     const uint8_t *end,
                                     level_type lvl)
{
  int32_t len;
  int level_bits = s->col.level_bits[lvl];
  int encoding   = lvl == level_type::DEFINITION ? s->page.definition_level_encoding
                                               : s->page.repetition_level_encoding;

  if (level_bits == 0) {
    len                       = 0;
    s->initial_rle_run[lvl]   = s->page.num_values * 2;  // repeated value
    s->initial_rle_value[lvl] = 0;
    s->lvl_start[lvl]         = cur;
  } else if (encoding == RLE) {
    if (cur + 4 < end) {
      uint32_t run;
      len = 4 + (cur[0]) + (cur[1] << 8) + (cur[2] << 16) + (cur[3] << 24);
      cur += 4;
      run                     = get_vlq32(cur, end);
      s->initial_rle_run[lvl] = run;
      if (!(run & 1)) {
        int v = (cur < end) ? cur[0] : 0;
        cur++;
        if (level_bits > 8) {
          v |= ((cur < end) ? cur[0] : 0) << 8;
          cur++;
        }
        s->initial_rle_value[lvl] = v;
      }
      s->lvl_start[lvl] = cur;
      if (cur > end) { s->error = 2; }
    } else {
      len      = 0;
      s->error = 2;
    }
  } else if (encoding == BIT_PACKED) {
    len                       = (s->page.num_values * level_bits + 7) >> 3;
    s->initial_rle_run[lvl]   = ((s->page.num_values + 7) >> 3) * 2 + 1;  // literal run
    s->initial_rle_value[lvl] = 0;
    s->lvl_start[lvl]         = cur;
  } else {
    s->error = 3;
    len      = 0;
  }
  return (uint32_t)len;
}

/**
 * @brief Decode definition or repetition levels
 *
 * @param[in,out] s Page state input/output
 * @param[in] t target_count Target count of non-NULL values on output
 * @param[in] t Warp0 thread ID (0..31)
 * @param[in] lvl The level type we are decoding - DEFINITION or REPETITION
 */
__device__ int32_t
gpuDecodeStream(uint32_t *output, page_state_s *s, int32_t count, int t, level_type lvl)
{
  const uint8_t *cur_def    = s->lvl_start[lvl];
  const uint8_t *end        = s->data_start;
  uint32_t level_run        = s->initial_rle_run[lvl];
  int32_t level_val         = s->initial_rle_value[lvl];
  int level_bits            = s->col.level_bits[lvl];
  int max_level             = s->col.max_level[lvl];
  int32_t num_values        = s->num_values;
  int32_t value_count       = s->lvl_count[lvl];
  int32_t batch_coded_count = 0;

  if (!t && s->col.col_index == column_watch) {
        //printf("GDS START %d, (%d) : decoding %d values, current count %d, %d\n", t, lvl, count,
        //value_count, max_level);
  }

  // while (batch_coded_count < count && value_count < num_values) {
  while (value_count < count && value_count < num_values) {
    int batch_len, is_valid;
    uint32_t valid_mask;
    if (level_run <= 1) {
      // Get a new run symbol from the byte stream
      int sym_len = 0;
      if (!t) {
        const uint8_t *cur = cur_def;
        if (cur < end) { level_run = get_vlq32(cur, end); }
        if (!(level_run & 1)) {
          if (cur < end) level_val = cur[0];
          cur++;
          if (level_bits > 8) {
            if (cur < end) level_val |= cur[0] << 8;
            cur++;
          }
        }
        if (cur > end || level_run <= 1) { s->error = 0x10; }
        sym_len = (int32_t)(cur - cur_def);
        __threadfence_block();
      }
      sym_len   = SHFL0(sym_len);
      level_val = SHFL0(level_val);
      level_run = SHFL0(level_run);
      cur_def += sym_len;
    }
    if (s->error) { break; }

    batch_len = min(num_values - value_count, 32);
    if (level_run & 1) {
      // Literal run
      int batch_len8;
      batch_len  = min(batch_len, (level_run >> 1) * 8);
      batch_len8 = (batch_len + 7) >> 3;
      if (t < batch_len) {
        int bitpos         = t * level_bits;
        const uint8_t *cur = cur_def + (bitpos >> 3);
        bitpos &= 7;
        if (cur < end) level_val = cur[0];
        cur++;
        if (level_bits > 8 - bitpos && cur < end) {
          level_val |= cur[0] << 8;
          cur++;
          if (level_bits > 16 - bitpos && cur < end) level_val |= cur[0] << 16;
        }
        level_val = (level_val >> bitpos) & ((1 << level_bits) - 1);
      }
      level_run -= batch_len8 * 2;
      cur_def += batch_len8 * level_bits;
    } else {
      // Repeated value
      batch_len = min(batch_len, level_run >> 1);
      level_run -= batch_len * 2;
    }
    if (t < batch_len) {
      int idx = value_count + t;
      output[idx & (NZ_BFRSZ - 1)] = level_val;
    }
    batch_coded_count += batch_len;
    value_count += batch_len;
  }

  // update the stream info
  __syncthreads();
  if (!t) {
    s->lvl_start[lvl]         = cur_def;
    s->initial_rle_run[lvl]   = level_run;
    s->initial_rle_value[lvl] = level_val;
    s->lvl_count[lvl]         = value_count;

    if (!t && s->col.col_index == column_watch) {
       //printf("GDS END (%d) : decoded %d values, final count %d\n", lvl, batch_coded_count,
       //value_count);
    }
  }

  // return how many values we actually processed
  return batch_coded_count;
}

/**
 * @brief Performs RLE decoding of dictionary indexes
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target index position in dict_idx buffer (may exceed this value by up to
 * 31)
 * @param[in] t Warp1 thread ID (0..31)
 *
 * @return The new output position
 */
__device__ int gpuDecodeDictionaryIndices(volatile page_state_s *s, int target_pos, int t)
{
  const uint8_t *end = s->data_end;
  int dict_bits      = s->dict_bits;
  int pos            = s->dict_pos;

  while (pos < target_pos) {
    int is_literal, batch_len;
    if (!t) {
      uint32_t run       = s->dict_run;
      const uint8_t *cur = s->data_start;
      if (run <= 1) {
        run = (cur < end) ? get_vlq32(cur, end) : 0;
        if (!(run & 1)) {
          // Repeated value
          int bytecnt = (dict_bits + 7) >> 3;
          if (cur + bytecnt <= end) {
            int32_t run_val = cur[0];
            if (bytecnt > 1) {
              run_val |= cur[1] << 8;
              if (bytecnt > 2) {
                run_val |= cur[2] << 16;
                if (bytecnt > 3) { run_val |= cur[3] << 24; }
              }
            }
            s->dict_val = run_val & ((1 << dict_bits) - 1);
          }
          cur += bytecnt;
        }
      }
      if (run & 1) {
        // Literal batch: must output a multiple of 8, except for the last batch
        int batch_len_div8;
        batch_len      = max(min(32, (int)(run >> 1) * 8), 1);
        batch_len_div8 = (batch_len + 7) >> 3;
        run -= batch_len_div8 * 2;
        cur += batch_len_div8 * dict_bits;
      } else {
        batch_len = max(min(32, (int)(run >> 1)), 1);
        run -= batch_len * 2;
      }
      s->dict_run   = run;
      s->data_start = cur;
      is_literal    = run & 1;
      __threadfence_block();
    }
    SYNCWARP();
    is_literal = SHFL0(is_literal);
    batch_len  = SHFL0(batch_len);
    if (t < batch_len) {
      int dict_idx = s->dict_val;
      if (is_literal) {
        int32_t ofs      = (t - ((batch_len + 7) & ~7)) * dict_bits;
        const uint8_t *p = s->data_start + (ofs >> 3);
        ofs &= 7;
        if (p < end) {
          uint32_t c = 8 - ofs;
          dict_idx   = (*p++) >> ofs;
          if (c < dict_bits && p < end) {
            dict_idx |= (*p++) << c;
            c += 8;
            if (c < dict_bits && p < end) {
              dict_idx |= (*p++) << c;
              c += 8;
              if (c < dict_bits && p < end) { dict_idx |= (*p++) << c; }
            }
          }
          dict_idx &= (1 << dict_bits) - 1;
        }
      }
      s->dict_idx[(pos + t) & (NZ_BFRSZ - 1)] = dict_idx;
    }
    pos += batch_len;
  }
  return pos;
}

/**
 * @brief Performs RLE decoding of dictionary indexes, for when dict_size=1
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target write position
 * @param[in] t Thread ID
 *
 * @return The new output position
 */
__device__ int gpuDecodeRleBooleans(volatile page_state_s *s, int target_pos, int t)
{
  const uint8_t *end = s->data_end;
  int pos            = s->dict_pos;

  while (pos < target_pos) {
    int is_literal, batch_len;
    if (!t) {
      uint32_t run       = s->dict_run;
      const uint8_t *cur = s->data_start;
      if (run <= 1) {
        run = (cur < end) ? get_vlq32(cur, end) : 0;
        if (!(run & 1)) {
          // Repeated value
          s->dict_val = (cur < end) ? cur[0] & 1 : 0;
          cur++;
        }
      }
      if (run & 1) {
        // Literal batch: must output a multiple of 8, except for the last batch
        int batch_len_div8;
        batch_len = max(min(32, (int)(run >> 1) * 8), 1);
        if (batch_len >= 8) { batch_len &= ~7; }
        batch_len_div8 = (batch_len + 7) >> 3;
        run -= batch_len_div8 * 2;
        cur += batch_len_div8;
      } else {
        batch_len = max(min(32, (int)(run >> 1)), 1);
        run -= batch_len * 2;
      }
      s->dict_run   = run;
      s->data_start = cur;
      is_literal    = run & 1;
      __threadfence_block();
    }
    SYNCWARP();
    is_literal = SHFL0(is_literal);
    batch_len  = SHFL0(batch_len);
    if (t < batch_len) {
      int dict_idx;
      if (is_literal) {
        int32_t ofs      = t - ((batch_len + 7) & ~7);
        const uint8_t *p = s->data_start + (ofs >> 3);
        dict_idx         = (p < end) ? (p[0] >> (ofs & 7u)) & 1 : 0;
      } else {
        dict_idx = s->dict_val;
      }
      s->dict_idx[(pos + t) & (NZ_BFRSZ - 1)] = dict_idx;
    }
    pos += batch_len;
  }
  return pos;
}

/**
 * @brief Parses the length and position of strings
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target output position
 * @param[in] t Thread ID
 *
 * @return The new output position
 */
__device__ void gpuInitStringDescriptors(volatile page_state_s *s, int target_pos, int t)
{
  int pos = s->dict_pos;
  // This step is purely serial
  if (!t) {
    const uint8_t *cur = s->data_start;
    int dict_size      = s->dict_size;
    int k              = s->dict_val;

    while (pos < target_pos) {
      int len;
      if (k + 4 <= dict_size) {
        len = (cur[k]) | (cur[k + 1] << 8) | (cur[k + 2] << 16) | (cur[k + 3] << 24);
        k += 4;
        if (k + len > dict_size) { len = 0; }
      } else {
        len = 0;
      }
      s->dict_idx[pos & (NZ_BFRSZ - 1)] = k;
      s->str_len[pos & (NZ_BFRSZ - 1)]  = len;
      k += len;
      pos++;
    }
    s->dict_val = k;
    __threadfence_block();
  }
}

/**
 * @brief Output a string descriptor
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dstv Pointer to row output data (string descriptor or 32-bit hash)
 */
inline __device__ void gpuOutputString(volatile page_state_s *s,
                                       int src_pos,
                                       void *dstv,
                                       int value_idx)
{
  const char *ptr = NULL;
  size_t len      = 0;

  if (s->dict_base) {
    // String dictionary
    uint32_t dict_pos =
      (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] * sizeof(nvstrdesc_s) : 0;
    if (dict_pos < (uint32_t)s->dict_size) {
      const nvstrdesc_s *src = reinterpret_cast<const nvstrdesc_s *>(s->dict_base + dict_pos);
      ptr                    = src->ptr;
      len                    = src->count;
    }
  } else {
    // Plain encoding
    uint32_t dict_pos = s->dict_idx[src_pos & (NZ_BFRSZ - 1)];
    if (dict_pos <= (uint32_t)s->dict_size) {
      ptr = reinterpret_cast<const char *>(s->data_start + dict_pos);
      len = s->str_len[src_pos & (NZ_BFRSZ - 1)];
    }
  }
  if (s->dtype_len == 4) {
    // Output hash
    *reinterpret_cast<uint32_t *>(dstv) = device_str2hash32(ptr, len);
  } else {
    // Output string descriptor
    nvstrdesc_s *dst = reinterpret_cast<nvstrdesc_s *>(dstv);
    dst->ptr         = ptr;
    dst->count       = len;
  }
}

/**
 * @brief Output a boolean
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
inline __device__ void gpuOutputBoolean(volatile page_state_s *s, int src_pos, uint8_t *dst)
{
  *dst = s->dict_idx[src_pos & (NZ_BFRSZ - 1)];
}

/**
 * @brief Store a 32-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 */
inline __device__ void gpuStoreOutput(uint32_t *dst,
                                      const uint8_t *src8,
                                      uint32_t dict_pos,
                                      uint32_t dict_size)
{
  // uint32_t tmp;
  // dst = &tmp;

  uint32_t bytebuf;
  unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos < dict_size) {
    bytebuf = *(const uint32_t *)(src8 + dict_pos);
    if (ofs) {
      uint32_t bytebufnext = *(const uint32_t *)(src8 + dict_pos + 4);
      bytebuf              = __funnelshift_r(bytebuf, bytebufnext, ofs);
    }
  } else {
    bytebuf = 0;
  }
  *dst = bytebuf;
}

/**
 * @brief Store a 64-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 */
inline __device__ void gpuStoreOutput(uint2 *dst,
                                      const uint8_t *src8,
                                      uint32_t dict_pos,
                                      uint32_t dict_size)
{
  uint2 v;
  unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos < dict_size) {
    v.x = *(const uint32_t *)(src8 + dict_pos + 0);
    v.y = *(const uint32_t *)(src8 + dict_pos + 4);
    if (ofs) {
      uint32_t next = *(const uint32_t *)(src8 + dict_pos + 8);
      v.x           = __funnelshift_r(v.x, v.y, ofs);
      v.y           = __funnelshift_r(v.y, next, ofs);
    }
  } else {
    v.x = v.y = 0;
  }
  *dst = v;
}

/**
 * @brief Convert an INT96 Spark timestamp to 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
inline __device__ void gpuOutputInt96Timestamp(volatile page_state_s *s, int src_pos, int64_t *dst)
{
  const uint8_t *src8;
  uint32_t dict_pos, dict_size = s->dict_size, ofs;
  int64_t ts;

  if (s->dict_base) {
    // Dictionary
    dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
    src8     = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    src8     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos + 4 < dict_size) {
    uint3 v;
    int64_t nanos, secs, days;
    v.x = *(const uint32_t *)(src8 + dict_pos + 0);
    v.y = *(const uint32_t *)(src8 + dict_pos + 4);
    v.z = *(const uint32_t *)(src8 + dict_pos + 8);
    if (ofs) {
      uint32_t next = *(const uint32_t *)(src8 + dict_pos + 12);
      v.x           = __funnelshift_r(v.x, v.y, ofs);
      v.y           = __funnelshift_r(v.y, v.z, ofs);
      v.z           = __funnelshift_r(v.z, next, ofs);
    }
    nanos = v.y;
    nanos <<= 32;
    nanos |= v.x;
    // Convert from Julian day at noon to UTC seconds
    days = static_cast<int32_t>(v.z);
    secs = (days - 2440588) *
           (24 * 60 * 60);  // TBD: Should be noon instead of midnight, but this matches pyarrow
    if (s->col.ts_clock_rate)
      ts = (secs * s->col.ts_clock_rate) +
           nanos / (1000000000 / s->col.ts_clock_rate);  // Output to desired clock rate
    else
      ts = (secs * 1000000000) + nanos;
  } else {
    ts = 0;
  }
  *dst = ts;
}

/**
 * @brief Output a 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
inline __device__ void gpuOutputInt64Timestamp(volatile page_state_s *s, int src_pos, int64_t *dst)
{
  const uint8_t *src8;
  uint32_t dict_pos, dict_size = s->dict_size, ofs;
  int64_t ts;

  if (s->dict_base) {
    // Dictionary
    dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
    src8     = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    src8     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos + 4 < dict_size) {
    uint2 v;
    int64_t val;
    int32_t ts_scale;
    v.x = *(const uint32_t *)(src8 + dict_pos + 0);
    v.y = *(const uint32_t *)(src8 + dict_pos + 4);
    if (ofs) {
      uint32_t next = *(const uint32_t *)(src8 + dict_pos + 8);
      v.x           = __funnelshift_r(v.x, v.y, ofs);
      v.y           = __funnelshift_r(v.y, next, ofs);
    }
    val = v.y;
    val <<= 32;
    val |= v.x;
    // Output to desired clock rate
    ts_scale = s->ts_scale;
    if (ts_scale < 0) {
      // round towards negative infinity
      int sign = (val < 0);
      ts       = ((val + sign) / -ts_scale) + sign;
    } else {
      ts = val * ts_scale;
    }
  } else {
    ts = 0;
  }
  *dst = ts;
}

/**
 * @brief Powers of 10
 */
static const __device__ __constant__ double kPow10[40] = {
  1.0,   1.e1,  1.e2,  1.e3,  1.e4,  1.e5,  1.e6,  1.e7,  1.e8,  1.e9,  1.e10, 1.e11, 1.e12, 1.e13,
  1.e14, 1.e15, 1.e16, 1.e17, 1.e18, 1.e19, 1.e20, 1.e21, 1.e22, 1.e23, 1.e24, 1.e25, 1.e26, 1.e27,
  1.e28, 1.e29, 1.e30, 1.e31, 1.e32, 1.e33, 1.e34, 1.e35, 1.e36, 1.e37, 1.e38, 1.e39,
};

/**
 * @brief Output a decimal type ([INT32..INT128] + scale) as a 64-bit float
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 * @param[in] dtype Stored data type
 */
inline __device__ void gpuOutputDecimal(volatile page_state_s *s,
                                        int src_pos,
                                        double *dst,
                                        int dtype)
{
  const uint8_t *dict;
  uint32_t dict_pos, dict_size = s->dict_size, dtype_len_in;
  int64_t i128_hi, i128_lo;
  int32_t scale;
  double d;

  if (s->dict_base) {
    // Dictionary
    dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
    dict     = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    dict     = s->data_start;
  }
  dtype_len_in = s->dtype_len_in;
  dict_pos *= dtype_len_in;
  // FIXME: Not very efficient (currently reading 1 byte at a time) -> need a variable-length
  // unaligned load utility function (both little-endian and big-endian versions)
  if (dtype == INT32) {
    int32_t lo32 = 0;
    for (unsigned int i = 0; i < dtype_len_in; i++) {
      uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
      lo32 |= v << (i * 8);
    }
    i128_lo = lo32;
    i128_hi = lo32 >> 31;
  } else if (dtype == INT64) {
    int64_t lo64 = 0;
    for (unsigned int i = 0; i < dtype_len_in; i++) {
      uint64_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
      lo64 |= v << (i * 8);
    }
    i128_lo = lo64;
    i128_hi = lo64 >> 63;
  } else  // if (dtype == FIXED_LENGTH_BYTE_ARRAY)
  {
    i128_lo = 0;
    for (unsigned int i = dtype_len_in - min(dtype_len_in, 8); i < dtype_len_in; i++) {
      uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
      i128_lo    = (i128_lo << 8) | v;
    }
    if (dtype_len_in > 8) {
      i128_hi = 0;
      for (unsigned int i = dtype_len_in - min(dtype_len_in, 16); i < dtype_len_in - 8; i++) {
        uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
        i128_hi    = (i128_hi << 8) | v;
      }
      if (dtype_len_in < 16) {
        i128_hi <<= 64 - (dtype_len_in - 8) * 8;
        i128_hi >>= 64 - (dtype_len_in - 8) * 8;
      }
    } else {
      if (dtype_len_in < 8) {
        i128_lo <<= 64 - dtype_len_in * 8;
        i128_lo >>= 64 - dtype_len_in * 8;
      }
      i128_hi = i128_lo >> 63;
    }
  }
  scale = s->col.decimal_scale;
  d     = Int128ToDouble_rn(i128_lo, i128_hi);
  *dst  = (scale < 0) ? (d * kPow10[min(-scale, 39)]) : (d / kPow10[min(scale, 39)]);
}

/**
 * @brief Output a small fixed-length value
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
template <typename T>
inline __device__ void gpuOutputFast(volatile page_state_s *s, int src_pos, T *dst)
{
  const uint8_t *dict;
  uint32_t dict_pos, dict_size = s->dict_size;

  if (s->dict_base) {
    // Dictionary
    dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
    dict     = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    dict     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  gpuStoreOutput(dst, dict, dict_pos, dict_size);
}

/**
 * @brief Output a N-byte value
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst8 Pointer to row output data
 * @param[in] len Length of element
 */
static __device__ void gpuOutputGeneric(volatile page_state_s *s,
                                        int src_pos,
                                        uint8_t *dst8,
                                        int len)
{
  const uint8_t *dict;
  uint32_t dict_pos, dict_size = s->dict_size;

  if (s->dict_base) {
    // Dictionary
    dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
    dict     = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    dict     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  if (len & 3) {
    // Generic slow path
    for (unsigned int i = 0; i < len; i++) {
      dst8[i] = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
    }
  } else {
    // Copy 4 bytes at a time
    const uint8_t *src8 = dict;
    unsigned int ofs    = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;  // align to 32-bit boundary
    ofs <<= 3;    // bytes -> bits
    for (unsigned int i = 0; i < len; i += 4) {
      uint32_t bytebuf;
      if (dict_pos < dict_size) {
        bytebuf = *(const uint32_t *)(src8 + dict_pos);
        if (ofs) {
          uint32_t bytebufnext = *(const uint32_t *)(src8 + dict_pos + 4);
          bytebuf              = __funnelshift_r(bytebuf, bytebufnext, ofs);
        }
      } else {
        bytebuf = 0;
      }
      dict_pos += 4;
      *(uint32_t *)(dst8 + i) = bytebuf;
    }
  }
}

static __device__ bool setupLocalPageInfo(page_state_s *const s,
                                          PageInfo *pages,
                                          ColumnChunkDesc *chunks,
                                          size_t min_row,
                                          size_t num_rows,
                                          int32_t num_chunks)
{
  int page_idx = blockIdx.x;
  int t        = threadIdx.x;
  int chunk_idx;

  // Fetch page info
  // NOTE: Assumes that sizeof(PageInfo) <= 256 (and is padded to 4 bytes)
  if (t < sizeof(PageInfo) / sizeof(uint32_t)) {
    ((uint32_t *)&s->page)[t] = ((const uint32_t *)&pages[page_idx])[t];
  }
  __syncthreads();  
  if (s->page.flags & PAGEINFO_FLAGS_DICTIONARY) { return false; } 
  // Fetch column chunk info
  chunk_idx = s->page.chunk_idx;
  if ((uint32_t)chunk_idx < (uint32_t)num_chunks) {
    // NOTE: Assumes that sizeof(ColumnChunkDesc) <= 256 (and is padded to 4 bytes)
    if (t < sizeof(ColumnChunkDesc) / sizeof(uint32_t)) {
      ((uint32_t *)&s->col)[t] = ((const uint32_t *)&chunks[chunk_idx])[t];
    }
  }

  if (!t && s->col.col_index == column_watch) {
    /*
    printf(
      "PAGE SETUP START : page(%d), chunk_idx(%d/%d), col(%d), num_rows(%d), num_values(%d), "
      "chunk_row(%d), start_row(%d)\n",
      page_idx,
      chunk_idx,
      num_chunks,
      s->col.col_index,
      s->page.num_rows,
      s->page.num_values,
      s->page.chunk_row,
      s->col.start_row);
      */
  }

  // zero nested value and valid counts
  if(!t){
    printf("MND : %d\n", s->page.max_nesting_depth);
  }
  if (t < s->page.max_nesting_depth) {    
    s->page.nesting[t].o_value_count = 0;        
    s->page.nesting[t].o_valid_count = 0;
  }
  __syncthreads();

  if (!t) {
    s->error = 0;
    if (s->page.num_values > 0 && s->page.num_rows > 0) {
      uint8_t *cur          = s->page.page_data;
      uint8_t *end          = cur + s->page.uncompressed_page_size;      

      uint32_t dtype_len_out = s->col.data_type >> 3;
      s->ts_scale            = 0;
      // Validate data type
      switch (s->col.data_type & 7) {
        case BOOLEAN:
          s->dtype_len = 1;  // Boolean are stored as 1 byte on the output
          break;
        case INT32:
        case FLOAT: s->dtype_len = 4; break;
        case INT64:
          if (s->col.ts_clock_rate) {
            int32_t units = 0;
            if (s->col.converted_type == TIME_MICROS || s->col.converted_type == TIMESTAMP_MICROS)
              units = 1000000;
            else if (s->col.converted_type == TIME_MILLIS ||
                     s->col.converted_type == TIMESTAMP_MILLIS)
              units = 1000;
            if (units && units != s->col.ts_clock_rate)
              s->ts_scale = (s->col.ts_clock_rate < units) ? -(units / s->col.ts_clock_rate)
                                                           : (s->col.ts_clock_rate / units);
          }
          // Fall through to DOUBLE
        case DOUBLE: s->dtype_len = 8; break;
        case INT96: s->dtype_len = 12; break;
        case BYTE_ARRAY: s->dtype_len = sizeof(nvstrdesc_s); break;
        default:  // FIXED_LEN_BYTE_ARRAY:
          s->dtype_len = dtype_len_out;
          s->error |= (s->dtype_len <= 0);
          break;
      }
      // Special check for downconversions
      s->dtype_len_in = s->dtype_len;
      if (s->col.converted_type == DECIMAL) {
        s->dtype_len = 8;  // Convert DECIMAL to 64-bit float
      } else if ((s->col.data_type & 7) == INT32) {
        if (dtype_len_out == 1) s->dtype_len = 1;  // INT8 output
        if (dtype_len_out == 2) s->dtype_len = 2;  // INT16 output
      } else if ((s->col.data_type & 7) == BYTE_ARRAY && dtype_len_out == 4) {
        s->dtype_len = 4;  // HASH32 output
      } else if ((s->col.data_type & 7) == INT96) {
        s->dtype_len = 8;  // Convert to 64-bit timestamp
      }

      // our starting row (absolute index) is
      // col.start_row == absolute row index
      // page.chunk-row == relative row index within the chunk
      size_t page_start_row = s->col.start_row + s->page.chunk_row;
      // printf("PSR : %lu %d %lu\n", s->col.start_row, s->page.chunk_row, page_start_row);

      // during the decoding step we need to offset the global output buffers
      // for each level of nesting so that we write to the section this page
      // is reponsible for.  
      // - for flat schemas, we can do this directly by using row counts
      // - for nested schemas, these offsets are computed during the preprocess step
      if (s->col.column_data_base != nullptr) {                
        int max_depth   = s->col.max_level[level_type::REPETITION];
        for (int idx = 0; idx <= max_depth; idx++) {          
          int output_offset = max_depth == 0 ? page_start_row : s->page.nesting[idx].o_page_start_value;
          if(s->col.col_index == column_watch){
            //printf("PSV : %d\n", output_offset);
          }
          s->page.nesting[idx].data_out         = reinterpret_cast<uint8_t *>(s->col.column_data_base[idx]) + (output_offset * s->dtype_len);          
          s->page.nesting[idx].valid_map        = s->col.valid_map_base[idx];
          if(s->page.nesting[idx].valid_map != nullptr){
            s->page.nesting[idx].valid_map        += output_offset >> 5;
            s->page.nesting[idx].o_valid_map_offset = (int32_t)(output_offset & 0x1f);
          }          

          //printf("PTR : %lu, %lu, %d\n", (uint64_t)s->page.nesting[idx].data_out, (uint64_t)s->page.nesting[idx].valid_map, s->page.nesting[idx].o_valid_map_offset);
        }
      }
      s->first_output_value = 0;

      // first row within the page to start reading
      if (page_start_row >= min_row) {
        s->first_row = 0;
      } else {
        s->first_row = (int32_t)min(min_row - page_start_row, (size_t)s->page.num_rows);
      }
      // # of rows within the page to read
      s->num_rows = s->page.num_rows;
      if (page_start_row + s->num_rows > min_row + num_rows) {
        s->num_rows = (int32_t)max((int64_t)(min_row + num_rows - page_start_row), INT64_C(0));
      }

      // Find the compressed size of repetition levels
      cur += InitLevelSection(s, cur, end, level_type::REPETITION);
      // Find the compressed size of definition levels
      cur += InitLevelSection(s, cur, end, level_type::DEFINITION);

      s->dict_bits = 0;
      s->dict_base = 0;
      s->dict_size = 0;
      switch (s->page.encoding) {
        case PLAIN_DICTIONARY:
        case RLE_DICTIONARY:
          // RLE-packed dictionary indices, first byte indicates index length in bits
          if (((s->col.data_type & 7) == BYTE_ARRAY) && (s->col.str_dict_index)) {
            // String dictionary: use index
            s->dict_base = reinterpret_cast<const uint8_t *>(s->col.str_dict_index);
            s->dict_size = s->col.page_info[0].num_values * sizeof(nvstrdesc_s);
          } else {
            s->dict_base =
              s->col.page_info[0].page_data;  // dictionary is always stored in the first page
            s->dict_size = s->col.page_info[0].uncompressed_page_size;
          }
          s->dict_run  = 0;
          s->dict_val  = 0;
          s->dict_bits = (cur < end) ? *cur++ : 0;
          if (s->dict_bits > 32 || !s->dict_base) { s->error = (10 << 8) | s->dict_bits; }
          break;
        case PLAIN:
          s->dict_size = static_cast<int32_t>(end - cur);
          s->dict_val  = 0;
          if ((s->col.data_type & 7) == BOOLEAN) { s->dict_run = s->dict_size * 2 + 1; }
          break;
        case RLE: s->dict_run = 0; break;
        default:
          s->error = 1;  // Unsupported encoding
          break;
      }
      if (cur > end) { s->error = 1; }
      s->data_start = cur;
      s->data_end   = end;
    } else {
      s->error = 1;
    }

    s->page_value_count                  = 0;
    s->lvl_count[level_type::REPETITION] = 0;
    s->lvl_count[level_type::DEFINITION] = 0;
    s->nz_count                          = 0;
    s->num_values                        = s->page.num_values;
    s->dict_pos                          = 0;
    s->out_pos                           = 0;

    if (s->col.col_index == column_watch)
     {
       /*
      printf(
        "PAGE SETUP END : page(%d), chunk_idx(%d/%d), col(%d), first_value(%d), "
        "data(%lu)\n",
        page_idx,
        chunk_idx,
        num_chunks,
        s->col.col_index,
        s->first_row,        
        (uint64_t)s->data_out[0]);
        */
    }

    __threadfence_block();
  }
  __syncthreads();

  return true;
}

// now we have to transform that into actual row size information
static __device__ void gpuUpdateValidityOffsetsAndRowIndices(int32_t target_value_count,
                                                             page_state_s *s,
                                                             int t,
                                                             int page_idx,
                                                             bool display)
{          
  // max nesting depth of the column
  int max_depth   = s->col.max_level[level_type::REPETITION];
  // how many (input) values we've processed in the page so far
  int page_value_count = s->page_value_count;  
    
  while(page_value_count < target_value_count){
    // determine the nesting bounds for this thread
    int start_depth = -1;
    int end_depth = -1;
    int d = -1;
    if(page_value_count + t < target_value_count){
      int index = rolling_index(page_value_count + t);
      int r = s->rep[index];    
      start_depth = r;
      d = s->def[index];  
      end_depth = s->page.nesting[d].d_remap;
    }

    // compute the count mask for the first level 
    uint32_t count_mask = BALLOT((0 >= start_depth && 0 <= end_depth) ? 1 : 0);
       
    // always walk from 0 to max_depth even if our start and end depths are different.
    // otherwise we'd have thread/warp synchronization issues on the BALLOT() call.
    for(int s_idx=0; s_idx<=max_depth; s_idx++){
      PageNestingInfo *pni = &s->page.nesting[s_idx];
      int in_range = (s_idx >= start_depth && s_idx <= end_depth) ? 1 : 0;
    
      // everything up to the max_def_level is a real value
      int is_valid = 0;
      if (d >= pni->o_max_def_level && in_range) {
        is_valid = 1;
      }

      // each thread in the warp will set bit T, giving us a mask that tells us total count
      uint32_t valid_count_mask = BALLOT(is_valid);
        
      // if this is the value column emit an index
      if(is_valid && s_idx == max_depth){        
        // Note : mask & ((1 << t) - 1) implies "the count for all threads before me"
        int idx = pni->o_valid_count + __popc(valid_count_mask & ((1 << t) - 1));
        int ofs = pni->o_value_count + __popc(count_mask & ((1 << t) - 1));
        s->nz_idx[rolling_index(idx)] = ofs;
      }

      // compute the count mask for the -next- nesting level. in the case of
      // nested schemas we need this value to generate an offset for this level
      uint32_t next_count_mask = (s_idx < max_depth) ? BALLOT((s_idx+1 >= start_depth && s_idx+1 <= end_depth) ? 1 : 0) : 0;

      // if we're -not- at a leaf column, this is a nested schema, so emit an offset
      if(in_range && s_idx < max_depth){        
        int idx = pni->o_value_count + __popc(count_mask & ((1 << t) - 1));
        (reinterpret_cast<int *>(pni->data_out))[idx] = s->page.nesting[s_idx+1].o_value_count + __popc(next_count_mask & ((1 << t)-1));
      }

      // increment count of valid values and total values
      if(!t){
        pni->o_valid_count += __popc(valid_count_mask);
        pni->o_value_count += __popc(count_mask);
      }

      // update count_mask for the next level down      
      count_mask = next_count_mask;
    }

    page_value_count += min(32, (target_value_count - page_value_count) );
    __syncwarp();    
  }
  
  if(!t){    
    s->nz_count = s->page.nesting[max_depth].o_valid_count;
    s->page_value_count = page_value_count;
  }    
  
  #if 0      
  int max_depth   = s->col.max_level[level_type::REPETITION];
  int value_count = s->page.nesting[max_depth].o_value_count;

  int page_value_count = s->page_value_count;  

  // TODO : make this actually parallel
  if (!t) {
    // index of the field that stores the actual leaf values we're generating indices for
    while (page_value_count < target_value_count) {
      int index = rolling_index(page_value_count);
      if (s->col.col_index == column_watch  && display) {
        //printf("A0\n");
      }
      int r = s->rep[index];
      int d = s->def[index];
      if (s->col.col_index == column_watch && display) {
        //printf("A1\n");
      }

      // repetition level effectively means "nesting depth", which directly corresponds to the
      // output column index. for flat schemas, this value will always be 0.
      int start_depth = r;
      int end_depth   = s->page.nesting[d].d_remap;

      if (s->col.col_index == column_watch && display) {
        //printf("A2\n");
      }

      if (s->col.col_index == column_watch && display) {
        //printf("R/D : %d %d\n", r, d);
      }

      // walk from the start depth to the max depth
      for (int s_idx = start_depth; s_idx <= end_depth; s_idx++) {
        PageNestingInfo *pni = &s->page.nesting[s_idx];
        // printf("pni : %lu\n", pni);

        if (s->col.col_index == column_watch && display) {
          //printf("C0 %d %lu\n", s_idx, s->col.nesting);
        }

        if (d >= pni->o_max_def_level) {
          if (s->col.col_index == column_watch) {
            //            printf("C00 %lu\n", (uint64_t)s->valid_map);
            // printf("C01 %lu\n", (uint64_t)s->valid_map[s_idx]);
          }

          if (pni->valid_map != nullptr) {
            // multiple blocks are working on different pieces of the same column. hence the atomic.  of course, this
            // is the worst possible way to do this.  this is just here as a stopgap until this whole function becomes
            // parallel
            cudf::set_bit(pni->valid_map, pni->o_valid_count + pni->o_valid_map_offset);
          }

          // if we're at the leaf column, we've got a real non-null value
          if (s_idx == max_depth) { 
            s->nz_idx[rolling_index(s->nz_count++)] = value_count;          
          }
        }
        // everything after the end depth is null
        else {
          if (pni->valid_map != nullptr) {
            // multiple blocks are working on different pieces of the same column. hence the atomic.  of course, this
            // is the worst possible way to do this.  this is just here as a stopgap until this whole function becomes
            // parallel
            cudf::clear_bit(pni->valid_map,
                            pni->o_valid_count + pni->o_valid_map_offset);

            // multiple blocks are working on different pieces of the same column. hence the atomic.  of course, this
            // is the worst possible way to do this.  this is just here as a stopgap until this whole function becomes
            // parallel
            pni->o_null_count++;
          }
        }

        if (s_idx == max_depth) { value_count++; }

        // offsets
        if (s_idx != max_depth) {
          (reinterpret_cast<int *>(pni->data_out))[pni->o_value_count] = 
              s->page.nesting[s_idx+1].o_value_count + s->page.nesting[s_idx+1].o_page_start_value;
        }

        pni->o_valid_count++;
        pni->o_value_count++;
      }

      page_value_count++;
    }

    // update page value count
    s->page_value_count = page_value_count;
  
    /*
    if(s->col.col_index == column_watch){
      for(int idx=0; idx<=s->col.max_level[level_type::REPETITION]; idx++){
        printf("null_count[%d] %d\n", idx, s->col.nesting[idx].o_null_count);
        printf("valids[%d]", idx);
        for(int s_idx=0; s_idx<s->col.nesting[idx].o_size; s_idx++){
          printf("%d", bit_is_set(s->valid_map[idx], s_idx) ? 1 : 0);
        }
        printf("\n");
      }

      for(int idx=0; idx<s->col.max_level[level_type::REPETITION]; idx++){
        printf("offsets[%d] ", idx);
        for(int s_idx=0; s_idx<s->col.nesting[idx].o_size; s_idx++){
          printf("%d, ", (reinterpret_cast<int**>(s->data_out))[idx][s_idx]);
        }
        printf("\n");
      }
    }
    */       

    if (s->col.col_index == column_watch  && display) {
      //printf("GUV END : %d, %d %d\n", page_value_count, s->nz_count, s->num_values);
    }
  }
  #endif

  // if we're the terminating page for the column, add the final offset to each level of nesting
  if(!t){
    if (s->page_value_count >= s->num_values && s->page.flags & PAGEINFO_FLAGS_TERMINATOR) {
      for (int s_idx = 0; s_idx <= max_depth; s_idx++) {
        if (s_idx != max_depth) {
          (reinterpret_cast<int *>(s->page.nesting[s_idx].data_out))[s->page.nesting[s_idx].o_value_count] =
            s->page.nesting[s_idx+1].o_value_count + s->page.nesting[s_idx+1].o_page_start_value;
        }
      }
      printf("TERMINATED\n");
    }
  }  

  if(!t && s->col.col_index == column_watch){
    /*
    printf("vals : ");
    for(int idx=0; idx<s->nz_count; idx++){
      printf("%d, ", s->nz_idx[idx]);
    }
    printf("\n");
    */   
    for(int s_idx=0; s_idx<max_depth; s_idx++){
      printf("offsets(0) : ");
      for(int idx=0; idx<=s->page.nesting[s_idx].o_value_count; idx++){
        int offset = (reinterpret_cast<int *>(s->page.nesting[s_idx].data_out))[idx];
        printf("%d, ", offset);
      }
      printf("\n");
    }    
  }
}

// scan through repetition and definition levels, recording validity and
// (int the case of nested types) offset information, until we have produced
// at least target_count non-null values for the data decode step to process
__device__ void gpuDecodeLevels(page_state_s *s, int32_t target_count, int t, int page_idx)
{
  if (!t && s->col.col_index == column_watch) {
    //printf("GDL START, target_count : %d, current count : %d\n", target_count, s->nz_count);
  }
  
  int cur_target_count = target_count;
  while (!s->error && s->nz_count < target_count && s->page_value_count < s->num_values) {
    // decode repetition and definition levels
    gpuDecodeStream(s->rep, s, cur_target_count, t, level_type::REPETITION);
    gpuDecodeStream(s->def, s, cur_target_count, t, level_type::DEFINITION);
    __syncthreads();

    // because the rep and def streams can be encoded differently and we cannot request an exact
    // # of values to be decoded at once (it has to be a multiple of 8) we can only process
    // the lowest # of decoded rep/def levels we have.
    int target_value_count =
      min(s->lvl_count[level_type::REPETITION], s->lvl_count[level_type::DEFINITION]);
    
    // process as much as we can
    gpuUpdateValidityOffsetsAndRowIndices(target_value_count, s, t, page_idx, true);
    cur_target_count += 32;
    __syncthreads();
  }

  if (!t && s->col.col_index == column_watch) {
     //printf("GDL END, target_count : %d, current count : %d\n", target_count, s->nz_count);
  }
}

static __device__ void gpuUpdatePageSizes(page_state_s *s, int32_t target_value_count, int t)
{
  // max nesting depth of the column
  int max_depth   = s->col.max_level[level_type::REPETITION];
  // how many (input) values we've processed in the page so far
  int page_value_count = s->page_value_count;

  // TODO : make this actually parallel    
  /*
  if (!t && s->col.col_index == column_watch) {  
    printf("GUCS START : processing %d new values up to target %d (max depth : %d)\n",
            target_value_count - page_value_count,
            target_value_count,
            max_depth);            
  }  
  */
  while (page_value_count < target_value_count) {
    // determine the nesting bounds for this thread
    int start_depth = -1;
    int end_depth = -1;
    if(page_value_count + t < target_value_count){
      int index = rolling_index(page_value_count + t);
      int r = s->rep[index];    
      start_depth = r;
      int d = s->def[index];  
      end_depth   = s->page.nesting[d].d_remap;
    }

    // increment counts across all nesting depths
    for(int s_idx=0; s_idx<=max_depth; s_idx++){      
      int in_range = (s_idx >= start_depth && s_idx <= end_depth) ? 1 : 0;
      uint32_t count_mask = BALLOT(in_range);      
      if(!t){        
        s->page.nesting[s_idx].o_size += __popc(count_mask);
      }
    }
    page_value_count += min(32, (target_value_count - page_value_count) );
  }

  // update final page value count
  if(!t){
    s->page_value_count = target_value_count;
  }

  // if (s->col.col_index == column_watch) { printf("GUCS END : %d\n", page_value_count); }  

#if 0
  // TODO : make this actually parallel
  if (!t) {
    int max_depth   = s->col.max_level[level_type::REPETITION];
    int value_count = s->page.nesting[max_depth].o_value_count;

    int page_value_count = s->page_value_count;

    if (s->col.col_index == column_watch) {
      /*
      printf("GUCS START : processing %d new values up to target %d (max depth : %d)\n",
             target_value_count - page_value_count,
             target_value_count,
             max_depth);
             */
    }

    while (page_value_count < target_value_count) {
      int index = rolling_index(page_value_count);
      int r     = s->rep[index];
      int d     = s->def[index];

      /*
      if (r == 0) { 
        s->num_rows++;
      }
      */

      // walk from the start depth to the defined depth
      int start_depth = r;
      int end_depth   = s->page.nesting[d].d_remap;
      if (s->col.col_index == column_watch) {
//        printf("R/D : %d %d   S/E %d %d\n", r, d, start_depth, end_depth);
      }
      // multiple blocks are working on different pieces of the same column. hence the atomic.  of course, this
      // is the worst possible way to do this.  this is just here as a stopgap until this whole function becomes
      // parallel
      for (int s_idx = start_depth; s_idx <= end_depth; s_idx++) { 
        s->page.nesting[s_idx].o_size++;
      }
      page_value_count++;
    }

    // update page value count
    s->page_value_count = page_value_count;

    //if (s->col.col_index == column_watch) { printf("GUCS END : %d\n", page_value_count); }
  }
  #endif
}

// blockDim {NTHREADS,1,1}
extern "C" __global__ void __launch_bounds__(NTHREADS)
  gpuComputePageSizes(PageInfo *pages, ColumnChunkDesc *chunks, int32_t num_chunks)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s *const s = &state_g;
  int page_idx          = blockIdx.x;
  int t                 = threadIdx.x;

  if (!setupLocalPageInfo(s, pages, chunks, -1, -1, num_chunks)) { return; }

  // setupLocalPageInfo will have filled in the incorrect num_rows computed during
  // header decoding (it will only be valid in the non-nested case), so reset it to
  // zero here
  if (!t) { s->num_rows = 0; }
  __syncthreads();

  if (!t && s->col.col_index == column_watch) { 
    printf("GCC START\n"); 
  }

  // optimization : if gpuDecodeStream were capable of operating with more than 32 threads, we 
  // could go wider here. 
  if (t < 32) {
    // decode in batches of 32. not sure it makes sense to go wider since the decode functions only
    // operate on 1 warp.
    constexpr int batch_size = 32;
    int target_count = batch_size;
    while (!s->error && s->page_value_count < s->num_values) {
      // decode repetition and definition levels. these will attempt to decode at
      // least as up to the target, but may decode a few more. 
      gpuDecodeStream(s->rep, s, target_count, t, level_type::REPETITION);
      gpuDecodeStream(s->def, s, target_count, t, level_type::DEFINITION);
      __syncthreads();

      // we may have decoded different amounts from each stream, so only process what we've been
      int target_value_count =
        min(s->lvl_count[level_type::REPETITION], s->lvl_count[level_type::DEFINITION]);

      // process as much as we can
      gpuUpdatePageSizes(s, target_value_count, t);
      target_count += batch_size;
      __syncthreads();
    }
  }
  // update # rows in the actual page
  if(!t){
    // s->page.num_rows = s->num_rows;    
    s->page.num_rows = s->page.nesting[0].o_size;
  }

  if (!t && s->col.col_index == column_watch) {    
    printf("GCC END\n");
    printf("num rows == %d %d\n", s->page.chunk_idx, s->page.num_rows);    
    int max_depth   = s->col.max_level[level_type::REPETITION];    
    printf("Column %d (max_depth : %d):\n", s->col.col_index, max_depth);    
    for (int idx = 0; idx <= max_depth; idx++) {
      printf("   col_size[%d] : %d\n", idx, s->page.nesting[idx].o_size);
    }        
  }
}

/**
 * @brief Kernel for reading the column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk. If necessary, additional
 * conversion will be performed to translate from the Parquet datatype to
 * desired output datatype (ex. 32-bit to 16-bit, string to hash).
 *
 * @param[in] pages List of pages
 * @param[in,out] chunks List of column chunks
 * @param[in] min_row crop all rows below min_row
 * @param[in] num_rows Maximum number of rows to read
 * @param[in] num_chunks Number of column chunks
 **/
// blockDim {NTHREADS,1,1}
extern "C" __global__ void __launch_bounds__(NTHREADS) gpuDecodePageData(PageInfo *pages,
                                                                         ColumnChunkDesc *chunks,
                                                                         size_t min_row,
                                                                         size_t num_rows,
                                                                         int32_t num_chunks,
                                                                         bool has_nesting)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s *const s = &state_g;
  int page_idx          = blockIdx.x;
  int t                 = threadIdx.x;
  int out_thread0;

  if (!setupLocalPageInfo(s, pages, chunks, min_row, num_rows, num_chunks)) { return; }

  if (!t) {
    //printf("GDP START : %d, %d, %d, %d, %d\n", page_idx, (int)min_row, (int)num_rows, num_chunks,
     //s->num_values);
  }

  if (s->dict_base) {
    out_thread0 = (s->dict_bits > 0) ? 64 : 32;
  } else {
    out_thread0 =
      ((s->col.data_type & 7) == BOOLEAN || (s->col.data_type & 7) == BYTE_ARRAY) ? 64 : 32;
  }

  while (!s->error && (s->page_value_count < s->num_values || s->out_pos < s->nz_count)) {
    int target_pos;
    int out_pos = s->out_pos;

    if (t < out_thread0) {
      target_pos =
        min(out_pos + 2 * (NTHREADS - out_thread0), s->nz_count + (NTHREADS - out_thread0));
    } else {
      target_pos = min(s->nz_count, out_pos + NTHREADS - out_thread0);
      if (out_thread0 > 32) { target_pos = min(target_pos, s->dict_pos); }
    }
    __syncthreads();
    if (t < 32) {
      // decode repetition and definition levels.
      // - update validity vectors
      // - updates offsets (for nested columns)
      // - produces non-NULL value indices in s->nz_idx for subsequent decoding
      gpuDecodeLevels(s, target_pos, t, page_idx);
    } else if (t < out_thread0) {
      // WARP1: Decode dictionary indices, booleans or string positions
      if (s->dict_base) {
        target_pos = gpuDecodeDictionaryIndices(s, target_pos, t & 0x1f);
      } else if ((s->col.data_type & 7) == BOOLEAN) {
        target_pos = gpuDecodeRleBooleans(s, target_pos, t & 0x1f);
      } else if ((s->col.data_type & 7) == BYTE_ARRAY) {
        gpuInitStringDescriptors(s, target_pos, t & 0x1f);
      }
      if (t == 32) { *(volatile int32_t *)&s->dict_pos = target_pos; }
    } else {      
      // WARP1..WARP3: Decode values
      int dtype = s->col.data_type & 7;
      out_pos += t - out_thread0;
      int output_value_idx = s->nz_idx[rolling_index(out_pos)];

      // nesting level that is storing actual leaf values
      int value_level_index = s->col.max_level[level_type::REPETITION];
      
      if (out_pos < target_pos && output_value_idx >= 0 && s->first_output_value + output_value_idx < s->num_values) {
        uint32_t dtype_len = s->dtype_len;
        uint8_t *dst       = s->page.nesting[value_level_index].data_out + (size_t)output_value_idx * dtype_len;
        if (dtype == BYTE_ARRAY)
          gpuOutputString(s, out_pos, dst, output_value_idx);
        else if (dtype == BOOLEAN)
          gpuOutputBoolean(s, out_pos, dst);
        else if (s->col.converted_type == DECIMAL)
          gpuOutputDecimal(s, out_pos, reinterpret_cast<double *>(dst), dtype);
        else if (dtype == INT96)
          gpuOutputInt96Timestamp(s, out_pos, reinterpret_cast<int64_t *>(dst));
        else if (dtype_len == 8) {
          if (s->ts_scale)
            gpuOutputInt64Timestamp(s, out_pos, reinterpret_cast<int64_t *>(dst));
          else
            gpuOutputFast(s, out_pos, reinterpret_cast<uint2 *>(dst));
        } else if (dtype_len == 4)
          gpuOutputFast(s, out_pos, reinterpret_cast<uint32_t *>(dst));
        else
          gpuOutputGeneric(s, out_pos, dst, dtype_len);
      }
      
      if (t == out_thread0) { *(volatile int32_t *)&s->out_pos = target_pos; }
    }
    __syncthreads();
  }
  __syncthreads();
  if (!t) {
    // Update the number of rows (after cropping to [min_row, min_row+num_rows-1]), and number of
    // valid values

    // ROW VALUE
    // pages[page_idx].num_rows    = s->num_rows - s->first_row;

    // printf("END NULLS : %d %d\n", s->col.nesting[0].null_count, s->col.nesting[1].null_count);
    // pages[page_idx].num_values = s->num_values - s->first_value;

    // pages[page_idx].valid_count = (s->error) ? -s->error : s->page.valid_count;

    if (!t) {
      //printf("GDP END : %d\n", page_idx);
    }
  }
}

// this seems generally useful.  couldn't hurt to see if there's a way to genericize this
// as a general output-to-a-specific-field iterator
struct chunk_row_output_iter {
  PageInfo *p;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type *;
  using reference         = size_type &;
  using iterator_category = thrust::output_device_iterator_tag;

  chunk_row_output_iter operator+ __host__ __device__(int i)
  {
    return chunk_row_output_iter{p + i};
  }
  chunk_row_output_iter operator[] __host__ __device__(int i)
  {
    return chunk_row_output_iter{p + i};
  }
  void operator++ __host__ __device__() { p++; }
  reference operator*__host__ __device__() { return p->chunk_row; }
  void operator= __host__ __device__(value_type v) { p->chunk_row = v; }
};

struct start_offset_output_iterator {
  PageInfo *p;  
  int col_index;
  int nesting_depth;
  int empty = 0;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type *;
  using reference         = size_type &;
  using iterator_category = thrust::output_device_iterator_tag;

  start_offset_output_iterator operator+ __host__ __device__(int i)
  {    
    return start_offset_output_iterator{p + i, col_index, nesting_depth};
  }
  start_offset_output_iterator operator[] __host__ __device__(int i)
  {
    return start_offset_output_iterator{p + i, col_index, nesting_depth};
  }
  void operator++ __host__ __device__() { p++; }  
  reference operator*__host__ __device__() {             
    if(p->column_idx != col_index || p->flags & PAGEINFO_FLAGS_DICTIONARY){
      return empty;
    }
    return p->nesting[nesting_depth].o_page_start_value;
  }  
  void operator= __host__ __device__(value_type v) {         
    if(p->column_idx == col_index && !(p->flags & PAGEINFO_FLAGS_DICTIONARY)){
      p->nesting[nesting_depth].o_page_start_value = v;
    }
  }
};

hipError_t PreprocessColumnData(hostdevice_vector<PageInfo>& pages,
                                 hostdevice_vector<ColumnChunkDesc>& chunks,
                                 std::vector<std::vector<std::pair<int, bool>>>& nested_info,
                                 size_t num_rows,
                                 size_t min_row,
                                 hipStream_t stream)
{
  dim3 dim_block(NTHREADS, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page

  // computes:
  // per-PageInfo value counts for all levels of nesting.
  // per-PageInfo # of rows
  gpuComputePageSizes<<<dim_grid, dim_block, 0, stream>>>(pages.device_ptr(), chunks.device_ptr(), chunks.size());  

  // computes:
  // output column sizes for each level of nesting
  // per-page start offsets for each level of nesting
  for(size_t idx=0; idx<nested_info.size(); idx++){
    size_t max_depth = nested_info[idx].size()-1;
    for(size_t l_idx=0; l_idx<=max_depth; l_idx++){      
      // column size
      auto page_input = thrust::make_transform_iterator(pages.device_ptr(), [idx, l_idx] __device__ (PageInfo const& page){
        if(page.column_idx != idx || page.flags & PAGEINFO_FLAGS_DICTIONARY){
          return 0;
        }
        return page.nesting[l_idx].o_size;
      });      
      nested_info[idx][l_idx].first = thrust::reduce(rmm::exec_policy(stream)->on(stream), page_input, page_input + pages.size());

      // add 1 for non-leaf levels for the terminating offset
      if(l_idx < max_depth){
        nested_info[idx][l_idx].first++;
      }

      // per-page start offset
      auto key_input = thrust::make_transform_iterator(pages.device_ptr(), [] __device__ (PageInfo const& page){
        return page.column_idx;      
      });
      thrust::exclusive_scan_by_key(rmm::exec_policy(stream)->on(stream),
                                  key_input,
                                  key_input + pages.size(),
                                  page_input,
                                  start_offset_output_iterator{pages.device_ptr(), static_cast<int>(chunks[idx].col_index), static_cast<int>(l_idx)});
    }    
  }
      
  // computes:
  // PageInfo::chunk_row for all pages
  // !!! This is making the assumption that ordering of pages is sorted by chunk_idx !!!
  auto key_input = thrust::make_transform_iterator(pages.device_ptr(), [] __device__ (PageInfo const& page){
    return page.chunk_idx;
  }); 
  auto page_input = thrust::make_transform_iterator(pages.device_ptr(), [] __device__ (PageInfo const& page){
    return page.num_rows;
  });
  thrust::exclusive_scan_by_key(rmm::exec_policy(stream)->on(stream),
                                key_input,
                                key_input + pages.size(),
                                page_input,
                                chunk_row_output_iter{pages.device_ptr()});    

  PageNestingInfo pni[16];   
  for(size_t idx=0; idx<nested_info.size(); idx++){
    int max_depth = nested_info[idx].size();
    printf("Column %lu info\n", idx);
    for(int l_idx=0; l_idx<max_depth; l_idx++){
      printf("   depth %d : size(%d), nullable(%s)\n", l_idx, nested_info[idx][l_idx].first, nested_info[idx][l_idx].second ? "yes" : "no");
    }
  }  
  for(size_t idx=0; idx<chunks.size(); idx++){
    int max_depth = chunks[idx].max_level[level_type::REPETITION] + 1;
    for(int p_idx=0; p_idx<chunks[idx].num_data_pages; p_idx++){
      PageInfo pi;
      hipMemcpy(&pi, chunks[idx].page_info + chunks[idx].num_dict_pages + p_idx, sizeof(PageInfo), hipMemcpyDeviceToHost);      
      hipMemcpy(pni, pi.nesting, sizeof(PageNestingInfo) * max_depth, hipMemcpyDeviceToHost);
      printf("Column %d, rg %lu, page %d (%d) (%s), num_values : %d, nested offsets:\n", chunks[idx].col_index, idx, p_idx, pi.column_idx, pi.flags & PAGEINFO_FLAGS_TERMINATOR ? "T" : "", pi.num_values);
      for(int l_idx=0; l_idx<max_depth; l_idx++){
        printf("   depth %d : %d\n", l_idx, pni[l_idx].o_page_start_value);
      }
    }    
  }

  return hipSuccess;
}

hipError_t __host__ DecodePageData(PageInfo *pages,
                                    int32_t num_pages,
                                    ColumnChunkDesc *chunks,
                                    int32_t num_chunks,
                                    size_t num_rows,
                                    size_t min_row,
                                    bool has_nesting,
                                    hipStream_t stream)
{
  dim3 dim_block(NTHREADS, 1);
  dim3 dim_grid(num_pages, 1);  // 1 threadblock per page

  gpuDecodePageData<<<dim_grid, dim_block, 0, stream>>>(
    pages, chunks, min_row, num_rows, num_chunks, has_nesting);

  return hipSuccess;
}

}  // namespace gpu
}  // namespace parquet
}  // namespace io
}  // namespace cudf